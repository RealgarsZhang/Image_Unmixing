#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2016

       @author Azzam Haidar
       @author Stan Tomov
       
       @generated from magmablas/zlatanh.cu, normal z -> d, Tue Jul 18 11:35:57 2017

*/

#include "magma.h"

// BLK_X and BLK_Y need to be equal for dlaset_q to deal with diag & offdiag
// when looping over super blocks.
// Formerly, BLK_X and BLK_Y could be different.
#define BLK_X 64
#define BLK_Y BLK_X

#define MAGMA_D_TANH(x) tanh(x)

/******************************************************************************/
/*
    Divides matrix into ceil( m/BLK_X ) x ceil( n/BLK_Y ) blocks.
    Each block has BLK_X threads.
    Each thread loops across one row, updating BLK_Y entries.

    Code similar to dlaset, dlacpy, dlag2s, clag2z, dgeadd.
*/
__global__
void dlatanh_kernel(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column */
    bool full = (iby + BLK_Y <= n);
    /* do only rows inside matrix */
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            // full block-column
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = MAGMA_D_TANH(dA[j*ldda]);
            }
        }
        else {
            // partial block-column
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = MAGMA_D_TANH(dA[j*ldda]);
            }
        }
    }
}


/******************************************************************************

    Purpose
    -------
    DLATANH initialized dB by taking tanh of the corresponding element of 
    matrix dA.
    
    Arguments
    ---------
    @param[in]
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix dA.  N >= 0.
    
    @param[in]
    dA      DOUBLE PRECISION array, dimension (LDDA,N)
            The M-by-N matrix dA.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    @param[out]
    dB      DOUBLE PRECISION array, dimension (LDDB,N)
            The M-by-N matrix dB.
            On exit, dB = .tanh(dA)
    
    @param[in]
    lddb    INTEGER
            The leading dimension of the array dB.  LDDB >= max(1,M).
    
    @ingroup magma_ml
*******************************************************************************/
extern "C" void
magmablas_dlatanh(
    magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr const dA, magma_int_t ldda,
    magmaDouble_ptr             dB, magma_int_t lddb)
{

    if ( m == 0 || n == 0 ) {
        return;
    }

    dim3 threads( BLK_X, 1, 1 );
    dim3 grid( magma_ceildiv( m, BLK_X ), magma_ceildiv( n, BLK_Y ), 1 );

    dlatanh_kernel
        <<< grid, threads, 0, NULL >>>
        ( m, n, dA, ldda, dB, lddb );
}

